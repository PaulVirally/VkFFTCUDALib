#include "include/vkfft_cuda.cuh"

VkFFTConfiguration* new_config(const pfUINT fftdim, const pfUINT buffer_dim, const pfUINT* size, const pfUINT* omit_dims, const pfUINT num_batches,
                               const pfUINT coalesced_memory, const pfUINT aimThreads, const pfUINT numSharedBanks,
                               const bool forward, const bool use_double_precision, const bool inplace) {
    VkFFTConfiguration* const config = new VkFFTConfiguration({});

    // FFT dimension config
    config->FFTdim = fftdim;
    for (size_t i = 0; i < VKFFT_MAX_FFT_DIMENSIONS; ++i) {
        config->size[i] = size[i]; // We guarantee in Julia that size is of size VKFFT_MAX_FFT_DIMENSIONS and is not a nullptr
        config->omitDimension[i] = omit_dims[i]; // We guarantee in Julia that omit_dims is of size VKFFT_MAX_FFT_DIMENSIONS and is not a nullptr
    }

    // Batching config
    config->numberBatches = num_batches;
    if (num_batches < 1) {
        config->numberBatches = 1;
    }

    // Device config
    hipDevice_t* const device = new hipDevice_t; // FIXME: Use a smart pointer?
    if (hipCtxGetDevice(device) != hipSuccess) {
        // TODO: Let the user know about the error
        delete device;
        return nullptr;
    }
    config->device = device;

    // Buffer allocation (the buffer is the work area for the FFT)
    config->doublePrecision = use_double_precision;
    pfUINT* const buffer_size = new pfUINT(1); // FIXME: Use a smart pointer?
    for (size_t i = 0; i < buffer_dim; ++i) { // Product of all the non-omitted dimensions
        if (omit_dims[i]) continue;
        *buffer_size *= size[i];
    }
    *buffer_size *= config->numberBatches;
    *buffer_size *= 2 * (use_double_precision ? sizeof(double) : sizeof(float)); // *2 because we need to store both real and imaginary parts.
    config->bufferSize = buffer_size;
    void** const buffer_ptr = new void*; // FIXME: Use a smart pointer?
    *buffer_ptr = reinterpret_cast<void*>(1); // A dummy value, it just can't be nullptr
    config->buffer = buffer_ptr;

    if (!inplace) {
        config->isInputFormatted = true;

        void** const output_buffer_ptr = new void*; // FIXME: Use a smart pointer?
        *output_buffer_ptr = reinterpret_cast<void*>(2); // Another dummy value, must be different from buffer_ptr
        config->buffer = output_buffer_ptr;
        config->inputBuffer = buffer_ptr;

        config->inputBufferSize = buffer_size;
    }

    // Optional optimization parameters
    if (coalesced_memory) config->coalescedMemory = coalesced_memory;
    if (aimThreads) config->aimThreads = aimThreads;
    if (numSharedBanks) config->numSharedBanks = numSharedBanks;

    // The julia bindings create separate plans for forward and backward transforms, so we don't need to generate both kernels here
    config->makeForwardPlanOnly = forward;
    config->makeInversePlanOnly = !forward;

    return config;
}

void delete_config(VkFFTConfiguration* config) {
    if (config == nullptr) {
        return;
    }
    delete config->device;
    delete config->bufferSize;
    delete config->buffer;

    if ((config->outputBuffer != nullptr) && (config->buffer != config->outputBuffer)) {
        delete config->outputBuffer;
    }
    if ((config->inputBuffer != nullptr) && (config->buffer != config->inputBuffer) && (config->outputBuffer != config->inputBuffer)) {
        delete config->inputBuffer;
    }
    if ((config->inputBufferSize != nullptr) && (config->inputBufferSize != config->bufferSize)) {
        delete config->inputBufferSize;
    }
    if ((config->outputBufferSize != nullptr) && (config->outputBufferSize != config->bufferSize) && (config->outputBufferSize != config->inputBufferSize)) {
        delete config->outputBufferSize;
    }

    delete config;
}

VkFFTApplication* new_app(const VkFFTConfiguration* const config, VkFFTResult* const res) {
    if (config == nullptr) {
        return nullptr;
    }
    VkFFTApplication* const app = new VkFFTApplication({});
    *res = initializeVkFFT(app, *config);

    if (res == nullptr) {
        // TODO: This really should be unreachable
        // TODO: Let the user know about the error
        delete app;
        return nullptr;
    }

    if (*res != VKFFT_SUCCESS) {
        // TODO: Let the user know about the error
        delete app;
        return nullptr;
    }

    return app;
}

void delete_app(VkFFTApplication* app) {
    if (app == nullptr) {
        return;
    }
    deleteVkFFT(app);
    delete app;
}

VkFFTResult fft(VkFFTApplication* app, void* input_buffer, void* output_buffer, int direction) {
    *(app->configuration.buffer) = output_buffer;
    *(app->configuration.inputBuffer) = input_buffer;
    *(app->configuration.outputBuffer) = output_buffer;

    VkFFTLaunchParams params = {};
    params.buffer = app->configuration.buffer;
    params.inputBuffer = app->configuration.inputBuffer;
    params.outputBuffer = app->configuration.outputBuffer;

    return VkFFTAppend(app, direction, &params);
}